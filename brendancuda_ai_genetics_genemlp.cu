#include "brendancuda_ai_genetics_genemlp.cuh"
#include "brendancuda_random_rngfunc.cuh"
#include "hip/hip_runtime.h"

template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T>::GeneMLP(std::pair<T*, size_t> Base, MLP::MLP<T> Intermediate) {
    base = Base;
    this->Intermediate = Intermediate;
}
template <typename T>
std::pair<T*, size_t> BrendanCUDA::AI::Genetics::GeneMLP<T>::Base() {
    return base;
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::Dispose() {
    hipFree(base.first);
    Intermediate.Dispose();
}
template <typename T>
std::pair<T*, size_t> BrendanCUDA::AI::Genetics::GeneMLP<T>::Run() {
    return Intermediate.Run(base.first);
}
template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T> BrendanCUDA::AI::Genetics::GeneMLP<T>::Clone() {
    T* b1;
#ifdef _DEBUG
    auto a =
#endif
    hipMalloc(&b1, base.second * sizeof(T));
    std::pair<T*, size_t> nb(b1, base.second);
    hipMemcpy(b1, base.first, base.second * sizeof(T), hipMemcpyDeviceToDevice);
    MLP::MLP<T> ni = Intermediate.Clone();
    return GeneMLP<T>(nb, ni);
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::Randomize(T Scalar, Random::rngWState<uint64_t> rng) {
    RandomizeArray(base.first, base.second, Scalar, rng);
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    Intermediate.Randomize(Scalar, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::Randomize(T Scalar, T LowerBound, T UpperBound, Random::rngWState<uint64_t> rng) {
    RandomizeArray(base.first, base.second, Scalar, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    Intermediate.Randomize(Scalar, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::Randomize(T Scalar_Base, T Scalar_Intermediate, Random::rngWState<uint64_t> rng) {
    RandomizeArray(base.first, base.second, Scalar_Base, rng);
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    Intermediate.Randomize(Scalar_Intermediate, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::Randomize(T Scalar_Base, T Scalar_Intermediate, T LowerBound, T UpperBound, Random::rngWState<uint64_t> rng) {
    RandomizeArray(base.first, base.second, Scalar_Base, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    Intermediate.Randomize(Scalar_Intermediate, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
}
template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T> BrendanCUDA::AI::Genetics::GeneMLP<T>::Reproduce(T Scalar, Random::rngWState<uint64_t> rng) {
    GeneMLP<T> n = Clone();
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    n.Randomize(Scalar, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
    return n;
}
template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T> BrendanCUDA::AI::Genetics::GeneMLP<T>::Reproduce(T Scalar, T LowerBound, T UpperBound, Random::rngWState<uint64_t> rng) {
    GeneMLP<T> n = Clone();
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    n.Randomize(Scalar, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
    return n;
}
template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T> BrendanCUDA::AI::Genetics::GeneMLP<T>::Reproduce(T Scalar_Base, T Scalar_Intermediate, Random::rngWState<uint64_t> rng) {
    GeneMLP<T> n = Clone();
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    n.Randomize(Scalar_Base, Scalar_Intermediate, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
    return n;
}
template <typename T>
BrendanCUDA::AI::Genetics::GeneMLP<T> BrendanCUDA::AI::Genetics::GeneMLP<T>::Reproduce(T Scalar_Base, T Scalar_Intermediate, T LowerBound, T UpperBound, Random::rngWState<uint64_t> rng) {
    GeneMLP<T> n = Clone();
#ifdef _DEBUG
    auto x = hipDeviceSynchronize();
#endif
    n.Randomize(Scalar_Base, Scalar_Intermediate, LowerBound, UpperBound, rng);
#ifdef _DEBUG
    auto y = hipDeviceSynchronize();
#endif
    return n;
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::ZeroOverwrite() {
    InitZeroArray(base.first, Intermediate.InputLength());
    Intermediate.ZeroOverwrite();
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::RandomOverwrite(Random::rngWState<uint64_t> rng) {
    InitRandomArray(base.first, Intermediate.InputLength(), rng);
    Intermediate.RandomOverwrite(rng);
}
template <typename T>
void BrendanCUDA::AI::Genetics::GeneMLP<T>::RandomOverwrite(T LowerBound, T UpperBound, Random::rngWState<uint64_t> rng) {
    InitRandomArray(base.first, Intermediate.InputLength(), LowerBound, UpperBound, rng);
    Intermediate.RandomOverwrite(LowerBound, UpperBound, rng);
}

template BrendanCUDA::AI::Genetics::GeneMLP<float>;
template BrendanCUDA::AI::Genetics::GeneMLP<double>;