#include "hip/hip_runtime.h"
#include "brendancuda_mathfuncs.h"

#include <cmath>

template <>
__host__ __device__ float BrendanCUDA::Math::sqrt<float>(float value) {
    return std::sqrt(value);
}
template <>
__host__ __device__ double BrendanCUDA::Math::sqrt<double>(double value) {
    return std::sqrt(value);
}
template <>
__host__ __device__ int32_t BrendanCUDA::Math::sqrt<int32_t>(int32_t value) {
    if (value < 0) {
        return -1;
    }
    else if (value < 2) {
        return value;
    }

    int32_t lower = 2;
    int32_t upper = value >> 1;

    do {
        int32_t mid = lower + ((upper - lower) >> 1);
        if (value >= mid * mid) {
            lower = mid;
        }
        else {
            upper = mid;
        }
    } while (upper > (lower + 1));

    return lower;
}
template <>
__host__ __device__ uint32_t BrendanCUDA::Math::sqrt<uint32_t>(uint32_t value) {
    if (value < 2) {
        return value;
    }

    uint32_t lower = 2;
    uint32_t upper = value >> 1;

    do {
        uint32_t mid = lower + ((upper - lower) >> 1);
        if (value >= mid * mid) {
            lower = mid;
        }
        else {
            upper = mid;
        }
    } while (upper > (lower + 1));

    return lower;
}
template <>
__host__ __device__ int8_t BrendanCUDA::Math::sqrt<int8_t>(int8_t value) {
    return (int8_t)sqrt((int32_t)value);
}
template <>
__host__ __device__ uint8_t BrendanCUDA::Math::sqrt<uint8_t>(uint8_t value) {
    return (uint8_t)sqrt((uint32_t)value);
}
template <>
__host__ __device__ int16_t BrendanCUDA::Math::sqrt<int16_t>(int16_t value) {
    return (int16_t)sqrt((int32_t)value);
}
template <>
__host__ __device__ uint16_t BrendanCUDA::Math::sqrt<uint16_t>(uint16_t value) {
    return (uint16_t)sqrt((uint32_t)value);
}
template <>
__host__ __device__ int64_t BrendanCUDA::Math::sqrt<int64_t>(int64_t value) {
    if (value < 0) {
        return -1;
    }
    else if (value < 2) {
        return value;
    }

    int64_t lower = 2;
    int64_t upper = value >> 1;

    do {
        int64_t mid = lower + ((upper - lower) >> 1);
        if (value >= mid * mid) {
            lower = mid;
        }
        else {
            upper = mid;
        }
    } while (upper > (lower + 1));

    return lower;
}
template <>
__host__ __device__ uint64_t BrendanCUDA::Math::sqrt<uint64_t>(uint64_t value) {
    if (value < 2) {
        return value;
    }

    uint64_t lower = 2;
    uint64_t upper = value >> 1;

    do {
        uint64_t mid = lower + ((upper - lower) >> 1);
        if (value >= mid * mid) {
            lower = mid;
        }
        else {
            upper = mid;
        }
    } while (upper > (lower + 1));

    return lower;
}

template <typename _T>
__host__ __device__ _T BrendanCUDA::Math::clamp<_T>(_T value, _T lower, _T upper) {
    if (value < lower) {
        return lower;
    }
    if (value > upper) {
        return upper;
    }
    return value;
}

template float BrendanCUDA::Math::sqrt<float>(float);
template double BrendanCUDA::Math::sqrt<double>(double);
template int8_t BrendanCUDA::Math::sqrt<int8_t>(int8_t);
template uint8_t BrendanCUDA::Math::sqrt<uint8_t>(uint8_t);
template int16_t BrendanCUDA::Math::sqrt<int16_t>(int16_t);
template uint16_t BrendanCUDA::Math::sqrt<uint16_t>(uint16_t);
template int32_t BrendanCUDA::Math::sqrt<int32_t>(int32_t);
template uint32_t BrendanCUDA::Math::sqrt<uint32_t>(uint32_t);
template int64_t BrendanCUDA::Math::sqrt<int64_t>(int64_t);
template uint64_t BrendanCUDA::Math::sqrt<uint64_t>(uint64_t);

template float BrendanCUDA::Math::clamp<float>(float, float, float);
template double BrendanCUDA::Math::clamp<double>(double, double, double);
template int8_t BrendanCUDA::Math::clamp<int8_t>(int8_t, int8_t, int8_t);
template uint8_t BrendanCUDA::Math::clamp<uint8_t>(uint8_t, uint8_t, uint8_t);
template int16_t BrendanCUDA::Math::clamp<int16_t>(int16_t, int16_t, int16_t);
template uint16_t BrendanCUDA::Math::clamp<uint16_t>(uint16_t, uint16_t, uint16_t);
template int32_t BrendanCUDA::Math::clamp<int32_t>(int32_t, int32_t, int32_t);
template uint32_t BrendanCUDA::Math::clamp<uint32_t>(uint32_t, uint32_t, uint32_t);
template int64_t BrendanCUDA::Math::clamp<int64_t>(int64_t, int64_t, int64_t);
template uint64_t BrendanCUDA::Math::clamp<uint64_t>(uint64_t, uint64_t, uint64_t);