#include "hip/hip_runtime.h"
#include "details_fillwith.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <>

__global__ void fillWithKernel(void* Array, void* Value, size_t ValueSize) {
    memcpy((uint8_t*)Array + blockIdx.x * ValueSize, Value, ValueSize);
}

namespace bcuda {
    namespace details {
        __forceinline void FillWith(void* Array, size_t ArrayElementCount, void* Value, size_t ValueSize) {
            fillWithKernel<<<ArrayElementCount, 1>>>(Array, Value, ValueSize);
        }
    }
}