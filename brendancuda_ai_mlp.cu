#include "hip/hip_runtime.h"
#include "brendancuda_ai_mlp.h"
#include "brendancuda_cudaerrorhelpers.h"
#include "brendancuda_devicecopy.cuh"

__global__ void runActivationFunctionOnArrayKernel(float* Array, BrendanCUDA::AI::activationFunction_t<float> ActivationFunction) {
    float& p(Array[blockIdx.x]);
    p = ActivationFunction(p);
}
__global__ void runActivationFunctionOnArrayKernel(double* Array, BrendanCUDA::AI::activationFunction_t<double> ActivationFunction) {
    double& p(Array[blockIdx.x]);
    p = ActivationFunction(p);
}
template <typename _T>
__host__ __device__ void BrendanCUDA::details::RunActivationFunctionOnArray(Span<_T> Array, AI::activationFunction_t<_T> ActivationFunction) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        _T& p(Array[i]);
        p = ActivationFunction(p);
    }
#else
    runActivationFunctionOnArrayKernel<<<Array.size, 1>>>(Array.ptr, ActivationFunction);
#endif
}

template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLP<_T>::MLP(size_t Length, activationFunction_t<_T> ActivationFunction) {
    len = Length;
    actnFunc = ActivationFunction;
#if __CUDA_ARCH__
    lyrs = new MLPL<_T>[Length];
#else
    ThrowIfBad(hipMalloc(&lyrs, Length * sizeof(MLPL<_T>)));
#endif
}
template <typename _T>
__host__ BrendanCUDA::AI::MLP::MLP<_T>::MLP(size_t Length, activationFunction_t<_T> ActivationFunction, MLPL<_T>* Layers, bool CopyFromHost)
    : MLP(Length, ActivationFunction) {
    ThrowIfBad(hipMemcpy(lyrs, Layers, Length * sizeof(MLPL<_T>), CopyFromHost ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}
template <typename _T>
__device__ BrendanCUDA::AI::MLP::MLP<_T>::MLP(size_t Length, activationFunction_t<_T> ActivationFunction, MLPL<_T>* Layers)
    : MLP(Length, ActivationFunction) {
    deviceMemcpy(lyrs, Layers, Length * sizeof(MLPL<_T>));
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::Dispose() {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->Dispose();
#else
        GetLayer(i).Dispose();
#endif
    }
#if __CUDA_ARCH__
    delete[] lyrs;
#else
    ThrowIfBad(hipFree(lyrs));
#endif
}
template <typename _T>
__host__ _T* BrendanCUDA::AI::MLP::MLP<_T>::Run(_T* Input) const {
    if (!len) {
        return 0;
    }
#if __CUDA_ARCH__
    MLPL<_T>* l = Layer(0);
    Input = l->Run(Input);
    details::RunActivationFunctionOnArray(Span<_T>(Input, l->OutputLength()), actnFunc);
#else
    MLPL<_T> l = GetLayer(0);
    Input = l.Run(Input);
    details::RunActivationFunctionOnArray(Span<_T>(Input, l.OutputLength()), actnFunc);
#endif
    for (size_t i = 1; i < len; ++i) {
#if __CUDA_ARCH__
        l = Layer(i);
        _T* nxt = l->Run(Input);
        details::RunActivationFunctionOnArray(Span<_T>(nxt, l->OutputLength()), actnFunc);
        delete[] Input;
#else
        l = GetLayer(i);
        _T* nxt = l.Run(Input);
        details::RunActivationFunctionOnArray(Span<_T>(nxt, l.OutputLength()), actnFunc);
        ThrowIfBad(hipFree(Input));
#endif
        Input = nxt;
    }
    return Input;
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLPL<_T>* BrendanCUDA::AI::MLP::MLP<_T>::Layers() const {
    return lyrs;
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLPL<_T>* BrendanCUDA::AI::MLP::MLP<_T>::Layer(size_t LayerIndex) const {
    return &lyrs[LayerIndex];
}
template <typename _T>
__host__ __device__ size_t BrendanCUDA::AI::MLP::MLP<_T>::LayerCount() const {
    return len;
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::activationFunction_t<_T> BrendanCUDA::AI::MLP::MLP<_T>::ActivationFunction() const {
    return actnFunc;
}
template <typename _T>
__host__ BrendanCUDA::AI::MLP::MLPL<_T>* BrendanCUDA::AI::MLP::MLP<_T>::GetLayers(bool CopyToHost) const {
    MLPL<_T>* p;
    if (CopyToHost) {
        p = new MLPL<_T>[len];
        ThrowIfBad(hipMemcpy(p, lyrs, sizeof(MLPL<_T>) * len, hipMemcpyDeviceToHost));
    }
    else {
        ThrowIfBad(hipMalloc(&p, sizeof(MLPL<_T>) * len));
        ThrowIfBad(hipMemcpy(p, lyrs, sizeof(MLPL<_T>) * len, hipMemcpyDeviceToDevice));
    }
    return p;
}
template <typename _T>
__device__ BrendanCUDA::AI::MLP::MLPL<_T>* BrendanCUDA::AI::MLP::MLP<_T>::GetLayers() const {
    MLPL<_T>* p = new MLPL<_T>[len];
    deviceMemcpy(p, lyrs, sizeof(MLPL<_T>) * len);
    return p;
}
template <typename _T>
__host__ void BrendanCUDA::AI::MLP::MLP<_T>::SetLayers(MLPL<_T>* Layers, bool CopyFromHost) {
    ThrowIfBad(hipMemcpy(lyrs, Layers, sizeof(MLPL<_T>) * len, CopyFromHost ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}
template <typename _T>
__device__ void BrendanCUDA::AI::MLP::MLP<_T>::SetLayers(MLPL<_T>* Layers) {
    deviceMemcpy(lyrs, Layers, sizeof(MLPL<_T>) * len);
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLPL<_T> BrendanCUDA::AI::MLP::MLP<_T>::GetLayer(size_t LayerIndex) const {
#if __CUDA_ARCH__
    return lyrs[LayerIndex];
#else
    MLPL<_T> r;
    ThrowIfBad(hipMemcpy(&r, &lyrs[LayerIndex], sizeof(MLPL<_T>), hipMemcpyDeviceToHost));
    return r;
#endif
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::SetLayer(size_t LayerIndex, MLPL<_T> Layer) {
#if __CUDA_ARCH__
    lyrs[LayerIndex] = Layer;
#else
    ThrowIfBad(hipMemcpy(&lyrs[LayerIndex], &Layer, sizeof(MLPL<_T>), hipMemcpyHostToDevice));
#endif
}
template <typename _T>
__host__ __device__ size_t BrendanCUDA::AI::MLP::MLP<_T>::InputLength() {
    if (len == 0) {
        return 0;
    }
#if __CUDA_ARCH__
    return Layer(0)->InputLength();
#else
    return GetLayer(0).InputLength();
#endif
}
template <typename _T>
__host__ __device__ size_t BrendanCUDA::AI::MLP::MLP<_T>::OutputLength() {
    if (len == 0) {
        return 0;
    }
#if __CUDA_ARCH__
    return Layer(len - 1)->OutputLength();
#else
    return GetLayer(len - 1).OutputLength();
#endif
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLP<_T> BrendanCUDA::AI::MLP::MLP<_T>::Clone() const {
    MLP<_T> m(len, actnFunc);
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        m.SetLayer(i, Layer(i)->Clone());
#else
        m.SetLayer(i, GetLayer(i).Clone());
#endif
    }
    return m;
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::Randomize(_T Scalar, Random::AnyRNG<uint64_t> RNG) {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->Randomize(Scalar, RNG);
#else
        GetLayer(i).Randomize(Scalar, RNG);
#endif
    }
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::Randomize(_T Scalar, _T LowerBound, _T UpperBound, Random::AnyRNG<uint64_t> RNG) {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->Randomize(Scalar, LowerBound, UpperBound, RNG);
#else
        GetLayer(i).Randomize(Scalar, LowerBound, UpperBound, RNG);
#endif
    }
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLP<_T> BrendanCUDA::AI::MLP::MLP<_T>::Reproduce(_T Scalar, Random::AnyRNG<uint64_t> RNG) const {
    MLP<_T> n = Clone();
    n.Randomize(Scalar, RNG);
    return n;
}
template <typename _T>
__host__ __device__ BrendanCUDA::AI::MLP::MLP<_T> BrendanCUDA::AI::MLP::MLP<_T>::Reproduce(_T Scalar, _T LowerBound, _T UpperBound, Random::AnyRNG<uint64_t> RNG) const {
    MLP<_T> n = Clone();
    n.Randomize(Scalar, LowerBound, UpperBound, RNG);
    return n;
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::ZeroOverwrite() {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->ZeroOverwrite();
#else
        GetLayer(i).ZeroOverwrite();
#endif
    }
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::RandomOverwrite(Random::AnyRNG<uint64_t> RNG) {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->RandomOverwrite(RNG);
#else
        GetLayer(i).RandomOverwrite(RNG);
#endif
    }
}
template <typename _T>
__host__ __device__ void BrendanCUDA::AI::MLP::MLP<_T>::RandomOverwrite(_T LowerBound, _T UpperBound, Random::AnyRNG<uint64_t> RNG) {
    for (size_t i = 0; i < len; ++i) {
#if __CUDA_ARCH__
        Layer(i)->RandomOverwrite(LowerBound, UpperBound, RNG);
#else
        GetLayer(i).RandomOverwrite(LowerBound, UpperBound, RNG);
#endif
    }
}
template <typename _T>
__host__ void BrendanCUDA::AI::MLP::MLP<_T>::Serialize(std::basic_ostream<char>& Stream) const {
    Stream.write((char*)&len, sizeof(size_t) / sizeof(char));

    for (size_t i = 0; i < len; ++i) {
        GetLayer(i).Serialize(Stream);
    }
}
template <typename _T>
__host__ BrendanCUDA::AI::MLP::MLP<_T> BrendanCUDA::AI::MLP::MLP<_T>::Deserialize(std::basic_istream<char>& Stream, activationFunction_t<_T> ActivationFunction) {
    size_t n_len;
    Stream.read((char*)&n_len, sizeof(size_t) / sizeof(char));

    MLPL<_T>* n_lyrs = new MLPL<_T>[n_len];

    for (size_t i = 0; i < n_len; ++i) {
        n_lyrs[i] = MLPL<_T>::Deserialize(Stream);
    }

    MLP<_T> mlp(n_len, ActivationFunction, n_lyrs, true);

    delete[] n_lyrs;

    return mlp;
}

template BrendanCUDA::AI::MLP::MLP<float>;
template BrendanCUDA::AI::MLP::MLP<double>;