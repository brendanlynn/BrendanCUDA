#include "hip/hip_runtime.h"
#include "brendancuda_ai.h"
#include "brendancuda_rand_bits.h"
#include "brendancuda_errorhelp.h"
#include "brendancuda_rand_drandom.cuh"
#include "brendancuda_rand_sseed.cuh"

using BrendanCUDA::Random::GetSeedOnKernel;
using BrendanCUDA::Random::DeviceRandom;
using BrendanCUDA::Random::HashI64;
using BrendanCUDA::Random::Get64Bits;
using BrendanCUDA::Random::AnyRNG;

template <typename _T>
__host__ __device__ _T BrendanCUDA::AI::ReLU(_T Value) {
    return (Value < (_T)0.) ? (_T)0. : Value;
}
template <typename _T>
__host__ __device__ _T BrendanCUDA::AI::TanH(_T Value) {
    return std::tanh(Value);
}
template <typename _T>
__host__ __device__ _T BrendanCUDA::AI::Sigmoid(_T Value) {
    Value = std::exp(Value);
    return Value / ((_T)1. + Value);
}

template float BrendanCUDA::AI::ReLU<float>(float);
template double BrendanCUDA::AI::ReLU<double>(double);
template float BrendanCUDA::AI::TanH<float>(float);
template double BrendanCUDA::AI::TanH<double>(double);
template float BrendanCUDA::AI::Sigmoid<float>(float);
template double BrendanCUDA::AI::Sigmoid<double>(double);

__global__ void randomizeArrayKernel(float* Array, float Scalar, uint64_t Seed) {
    float& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    float rd = (float)ts / (float)18446744073709551615;
    p += Scalar * (rd - 0.5f);
}
__global__ void randomizeArrayKernel(double* Array, double Scalar, uint64_t Seed) {
    double& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    double rd = (double)ts / (double)18446744073709551615;
    p += Scalar * (rd - 0.5);
}
__global__ void randomizeArrayKernel(float* Array, float Scalar, float LowerBound, float UpperBound, uint64_t Seed) {
    float& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    float rd = (float)ts / (float)18446744073709551615;
    float v = p + Scalar * (rd - 0.5f);
    if (v < LowerBound) {
        p = LowerBound;
}
    else if (v > UpperBound) {
        p = UpperBound;
    }
    else {
        p = v;
    }
}
__global__ void randomizeArrayKernel(double* Array, double Scalar, double LowerBound, double UpperBound, uint64_t Seed) {
    double& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    double rd = (double)ts / (double)18446744073709551615;
    double v = p + Scalar * (rd - 0.5);
    if (v < LowerBound) {
        p = LowerBound;
    }
    else if (v > UpperBound) {
        p = UpperBound;
    }
    else {
        p = v;
    }
}
__global__ void initRandomArrayKernel(float* Array, uint64_t Seed) {
    float& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    p = (float)ts / 18446744073709551615.f;
}
__global__ void initRandomArrayKernel(double* Array, uint64_t Seed) {
    double& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    p = (double)ts / 18446744073709551615.f;
}
__global__ void initRandomArrayKernel(uint64_t* Array, uint64_t Seed) {
    Array[blockIdx.x] = GetSeedOnKernel(Seed);
}
__global__ void initRandomArrayKernel(float* Array, float LowerBound, float Difference, uint64_t Seed) {
    constexpr float bs = 1.f / 18446744073709551615.f;

    float& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    p = (float)ts * bs * Difference + LowerBound;
}
__global__ void initRandomArrayKernel(double* Array, double LowerBound, double Difference, uint64_t Seed) {
    constexpr double bs = 1. / 18446744073709551615.;

    double& p(Array[blockIdx.x]);
    uint64_t ts = GetSeedOnKernel(Seed);
    p = (double)ts * bs * Difference + LowerBound;
}
__host__ __device__ void copyFloatsToInt32Func(float* Floats, uint32_t* Int32, float Split) {
    uint32_t m = 0;
    if (Floats[0] > Split) {
        m |= 1ui32 << 0;
    }
    if (Floats[1] > Split) {
        m |= 1ui32 << 1;
    }
    if (Floats[2] > Split) {
        m |= 1ui32 << 2;
    }
    if (Floats[3] > Split) {
        m |= 1ui32 << 3;
    }
    if (Floats[4] > Split) {
        m |= 1ui32 << 4;
    }
    if (Floats[5] > Split) {
        m |= 1ui32 << 5;
    }
    if (Floats[6] > Split) {
        m |= 1ui32 << 6;
    }
    if (Floats[7] > Split) {
        m |= 1ui32 << 7;
    }
    if (Floats[8] > Split) {
        m |= 1ui32 << 8;
    }
    if (Floats[9] > Split) {
        m |= 1ui32 << 9;
    }
    if (Floats[10] > Split) {
        m |= 1ui32 << 10;
    }
    if (Floats[11] > Split) {
        m |= 1ui32 << 11;
    }
    if (Floats[12] > Split) {
        m |= 1ui32 << 12;
    }
    if (Floats[13] > Split) {
        m |= 1ui32 << 13;
    }
    if (Floats[14] > Split) {
        m |= 1ui32 << 14;
    }
    if (Floats[15] > Split) {
        m |= 1ui32 << 15;
    }
    if (Floats[16] > Split) {
        m |= 1ui32 << 16;
    }
    if (Floats[17] > Split) {
        m |= 1ui32 << 17;
    }
    if (Floats[18] > Split) {
        m |= 1ui32 << 18;
    }
    if (Floats[19] > Split) {
        m |= 1ui32 << 19;
    }
    if (Floats[20] > Split) {
        m |= 1ui32 << 20;
    }
    if (Floats[21] > Split) {
        m |= 1ui32 << 21;
    }
    if (Floats[22] > Split) {
        m |= 1ui32 << 22;
    }
    if (Floats[23] > Split) {
        m |= 1ui32 << 23;
    }
    if (Floats[24] > Split) {
        m |= 1ui32 << 24;
    }
    if (Floats[25] > Split) {
        m |= 1ui32 << 25;
    }
    if (Floats[26] > Split) {
        m |= 1ui32 << 26;
    }
    if (Floats[27] > Split) {
        m |= 1ui32 << 27;
    }
    if (Floats[28] > Split) {
        m |= 1ui32 << 28;
    }
    if (Floats[29] > Split) {
        m |= 1ui32 << 29;
    }
    if (Floats[30] > Split) {
        m |= 1ui32 << 30;
    }
    if (Floats[31] > Split) {
        m |= 1ui32 << 31;
    }
    *Int32 = m;
}
__host__ __device__ void copyDoublesToInt32Func(double* Doubles, uint32_t* Int32, double Split) {
    uint32_t m = 0;
    if (Doubles[0] > Split) {
        m |= 1ui32 << 0;
    }
    if (Doubles[1] > Split) {
        m |= 1ui32 << 1;
    }
    if (Doubles[2] > Split) {
        m |= 1ui32 << 2;
    }
    if (Doubles[3] > Split) {
        m |= 1ui32 << 3;
    }
    if (Doubles[4] > Split) {
        m |= 1ui32 << 4;
    }
    if (Doubles[5] > Split) {
        m |= 1ui32 << 5;
    }
    if (Doubles[6] > Split) {
        m |= 1ui32 << 6;
    }
    if (Doubles[7] > Split) {
        m |= 1ui32 << 7;
    }
    if (Doubles[8] > Split) {
        m |= 1ui32 << 8;
    }
    if (Doubles[9] > Split) {
        m |= 1ui32 << 9;
    }
    if (Doubles[10] > Split) {
        m |= 1ui32 << 10;
    }
    if (Doubles[11] > Split) {
        m |= 1ui32 << 11;
    }
    if (Doubles[12] > Split) {
        m |= 1ui32 << 12;
    }
    if (Doubles[13] > Split) {
        m |= 1ui32 << 13;
    }
    if (Doubles[14] > Split) {
        m |= 1ui32 << 14;
    }
    if (Doubles[15] > Split) {
        m |= 1ui32 << 15;
    }
    if (Doubles[16] > Split) {
        m |= 1ui32 << 16;
    }
    if (Doubles[17] > Split) {
        m |= 1ui32 << 17;
    }
    if (Doubles[18] > Split) {
        m |= 1ui32 << 18;
    }
    if (Doubles[19] > Split) {
        m |= 1ui32 << 19;
    }
    if (Doubles[20] > Split) {
        m |= 1ui32 << 20;
    }
    if (Doubles[21] > Split) {
        m |= 1ui32 << 21;
    }
    if (Doubles[22] > Split) {
        m |= 1ui32 << 22;
    }
    if (Doubles[23] > Split) {
        m |= 1ui32 << 23;
    }
    if (Doubles[24] > Split) {
        m |= 1ui32 << 24;
    }
    if (Doubles[25] > Split) {
        m |= 1ui32 << 25;
    }
    if (Doubles[26] > Split) {
        m |= 1ui32 << 26;
    }
    if (Doubles[27] > Split) {
        m |= 1ui32 << 27;
    }
    if (Doubles[28] > Split) {
        m |= 1ui32 << 28;
    }
    if (Doubles[29] > Split) {
        m |= 1ui32 << 29;
    }
    if (Doubles[30] > Split) {
        m |= 1ui32 << 30;
    }
    if (Doubles[31] > Split) {
        m |= 1ui32 << 31;
    }
    *Int32 = m;
}
__host__ __device__ void copyFloatsToInt64Func(float* Floats, uint64_t* Int64, float Split) {
    uint64_t m = 0;
    if (Floats[0] > Split) {
        m |= 1ui64 << 0;
    }
    if (Floats[1] > Split) {
        m |= 1ui64 << 1;
    }
    if (Floats[2] > Split) {
        m |= 1ui64 << 2;
    }
    if (Floats[3] > Split) {
        m |= 1ui64 << 3;
    }
    if (Floats[4] > Split) {
        m |= 1ui64 << 4;
    }
    if (Floats[5] > Split) {
        m |= 1ui64 << 5;
    }
    if (Floats[6] > Split) {
        m |= 1ui64 << 6;
    }
    if (Floats[7] > Split) {
        m |= 1ui64 << 7;
    }
    if (Floats[8] > Split) {
        m |= 1ui64 << 8;
    }
    if (Floats[9] > Split) {
        m |= 1ui64 << 9;
    }
    if (Floats[10] > Split) {
        m |= 1ui64 << 10;
    }
    if (Floats[11] > Split) {
        m |= 1ui64 << 11;
    }
    if (Floats[12] > Split) {
        m |= 1ui64 << 12;
    }
    if (Floats[13] > Split) {
        m |= 1ui64 << 13;
    }
    if (Floats[14] > Split) {
        m |= 1ui64 << 14;
    }
    if (Floats[15] > Split) {
        m |= 1ui64 << 15;
    }
    if (Floats[16] > Split) {
        m |= 1ui64 << 16;
    }
    if (Floats[17] > Split) {
        m |= 1ui64 << 17;
    }
    if (Floats[18] > Split) {
        m |= 1ui64 << 18;
    }
    if (Floats[19] > Split) {
        m |= 1ui64 << 19;
    }
    if (Floats[20] > Split) {
        m |= 1ui64 << 20;
    }
    if (Floats[21] > Split) {
        m |= 1ui64 << 21;
    }
    if (Floats[22] > Split) {
        m |= 1ui64 << 22;
    }
    if (Floats[23] > Split) {
        m |= 1ui64 << 23;
    }
    if (Floats[24] > Split) {
        m |= 1ui64 << 24;
    }
    if (Floats[25] > Split) {
        m |= 1ui64 << 25;
    }
    if (Floats[26] > Split) {
        m |= 1ui64 << 26;
    }
    if (Floats[27] > Split) {
        m |= 1ui64 << 27;
    }
    if (Floats[28] > Split) {
        m |= 1ui64 << 28;
    }
    if (Floats[29] > Split) {
        m |= 1ui64 << 29;
    }
    if (Floats[30] > Split) {
        m |= 1ui64 << 30;
    }
    if (Floats[31] > Split) {
        m |= 1ui64 << 31;
    }
    if (Floats[32] > Split) {
        m |= 1ui64 << 32;
    }
    if (Floats[33] > Split) {
        m |= 1ui64 << 33;
    }
    if (Floats[34] > Split) {
        m |= 1ui64 << 34;
    }
    if (Floats[35] > Split) {
        m |= 1ui64 << 35;
    }
    if (Floats[36] > Split) {
        m |= 1ui64 << 36;
    }
    if (Floats[37] > Split) {
        m |= 1ui64 << 37;
    }
    if (Floats[38] > Split) {
        m |= 1ui64 << 38;
    }
    if (Floats[39] > Split) {
        m |= 1ui64 << 39;
    }
    if (Floats[40] > Split) {
        m |= 1ui64 << 40;
    }
    if (Floats[41] > Split) {
        m |= 1ui64 << 41;
    }
    if (Floats[42] > Split) {
        m |= 1ui64 << 42;
    }
    if (Floats[43] > Split) {
        m |= 1ui64 << 43;
    }
    if (Floats[44] > Split) {
        m |= 1ui64 << 44;
    }
    if (Floats[45] > Split) {
        m |= 1ui64 << 45;
    }
    if (Floats[46] > Split) {
        m |= 1ui64 << 46;
    }
    if (Floats[47] > Split) {
        m |= 1ui64 << 47;
    }
    if (Floats[48] > Split) {
        m |= 1ui64 << 48;
    }
    if (Floats[49] > Split) {
        m |= 1ui64 << 49;
    }
    if (Floats[50] > Split) {
        m |= 1ui64 << 50;
    }
    if (Floats[51] > Split) {
        m |= 1ui64 << 51;
    }
    if (Floats[52] > Split) {
        m |= 1ui64 << 52;
    }
    if (Floats[53] > Split) {
        m |= 1ui64 << 53;
    }
    if (Floats[54] > Split) {
        m |= 1ui64 << 54;
    }
    if (Floats[55] > Split) {
        m |= 1ui64 << 55;
    }
    if (Floats[56] > Split) {
        m |= 1ui64 << 56;
    }
    if (Floats[57] > Split) {
        m |= 1ui64 << 57;
    }
    if (Floats[58] > Split) {
        m |= 1ui64 << 58;
    }
    if (Floats[59] > Split) {
        m |= 1ui64 << 59;
    }
    if (Floats[60] > Split) {
        m |= 1ui64 << 60;
    }
    if (Floats[61] > Split) {
        m |= 1ui64 << 61;
    }
    if (Floats[62] > Split) {
        m |= 1ui64 << 62;
    }
    if (Floats[63] > Split) {
        m |= 1ui64 << 63;
    }
    *Int64 = m;
}
__host__ __device__ void copyDoublesToInt64Func(double* Doubles, uint64_t* Int64, double Split) {
    uint64_t m = 0;
    if (Doubles[0] > Split) {
        m |= 1ui64 << 0;
    }
    if (Doubles[1] > Split) {
        m |= 1ui64 << 1;
    }
    if (Doubles[2] > Split) {
        m |= 1ui64 << 2;
    }
    if (Doubles[3] > Split) {
        m |= 1ui64 << 3;
    }
    if (Doubles[4] > Split) {
        m |= 1ui64 << 4;
    }
    if (Doubles[5] > Split) {
        m |= 1ui64 << 5;
    }
    if (Doubles[6] > Split) {
        m |= 1ui64 << 6;
    }
    if (Doubles[7] > Split) {
        m |= 1ui64 << 7;
    }
    if (Doubles[8] > Split) {
        m |= 1ui64 << 8;
    }
    if (Doubles[9] > Split) {
        m |= 1ui64 << 9;
    }
    if (Doubles[10] > Split) {
        m |= 1ui64 << 10;
    }
    if (Doubles[11] > Split) {
        m |= 1ui64 << 11;
    }
    if (Doubles[12] > Split) {
        m |= 1ui64 << 12;
    }
    if (Doubles[13] > Split) {
        m |= 1ui64 << 13;
    }
    if (Doubles[14] > Split) {
        m |= 1ui64 << 14;
    }
    if (Doubles[15] > Split) {
        m |= 1ui64 << 15;
    }
    if (Doubles[16] > Split) {
        m |= 1ui64 << 16;
    }
    if (Doubles[17] > Split) {
        m |= 1ui64 << 17;
    }
    if (Doubles[18] > Split) {
        m |= 1ui64 << 18;
    }
    if (Doubles[19] > Split) {
        m |= 1ui64 << 19;
    }
    if (Doubles[20] > Split) {
        m |= 1ui64 << 20;
    }
    if (Doubles[21] > Split) {
        m |= 1ui64 << 21;
    }
    if (Doubles[22] > Split) {
        m |= 1ui64 << 22;
    }
    if (Doubles[23] > Split) {
        m |= 1ui64 << 23;
    }
    if (Doubles[24] > Split) {
        m |= 1ui64 << 24;
    }
    if (Doubles[25] > Split) {
        m |= 1ui64 << 25;
    }
    if (Doubles[26] > Split) {
        m |= 1ui64 << 26;
    }
    if (Doubles[27] > Split) {
        m |= 1ui64 << 27;
    }
    if (Doubles[28] > Split) {
        m |= 1ui64 << 28;
    }
    if (Doubles[29] > Split) {
        m |= 1ui64 << 29;
    }
    if (Doubles[30] > Split) {
        m |= 1ui64 << 30;
    }
    if (Doubles[31] > Split) {
        m |= 1ui64 << 31;
    }
    if (Doubles[32] > Split) {
        m |= 1ui64 << 32;
    }
    if (Doubles[33] > Split) {
        m |= 1ui64 << 33;
    }
    if (Doubles[34] > Split) {
        m |= 1ui64 << 34;
    }
    if (Doubles[35] > Split) {
        m |= 1ui64 << 35;
    }
    if (Doubles[36] > Split) {
        m |= 1ui64 << 36;
    }
    if (Doubles[37] > Split) {
        m |= 1ui64 << 37;
    }
    if (Doubles[38] > Split) {
        m |= 1ui64 << 38;
    }
    if (Doubles[39] > Split) {
        m |= 1ui64 << 39;
    }
    if (Doubles[40] > Split) {
        m |= 1ui64 << 40;
    }
    if (Doubles[41] > Split) {
        m |= 1ui64 << 41;
    }
    if (Doubles[42] > Split) {
        m |= 1ui64 << 42;
    }
    if (Doubles[43] > Split) {
        m |= 1ui64 << 43;
    }
    if (Doubles[44] > Split) {
        m |= 1ui64 << 44;
    }
    if (Doubles[45] > Split) {
        m |= 1ui64 << 45;
    }
    if (Doubles[46] > Split) {
        m |= 1ui64 << 46;
    }
    if (Doubles[47] > Split) {
        m |= 1ui64 << 47;
    }
    if (Doubles[48] > Split) {
        m |= 1ui64 << 48;
    }
    if (Doubles[49] > Split) {
        m |= 1ui64 << 49;
    }
    if (Doubles[50] > Split) {
        m |= 1ui64 << 50;
    }
    if (Doubles[51] > Split) {
        m |= 1ui64 << 51;
    }
    if (Doubles[52] > Split) {
        m |= 1ui64 << 52;
    }
    if (Doubles[53] > Split) {
        m |= 1ui64 << 53;
    }
    if (Doubles[54] > Split) {
        m |= 1ui64 << 54;
    }
    if (Doubles[55] > Split) {
        m |= 1ui64 << 55;
    }
    if (Doubles[56] > Split) {
        m |= 1ui64 << 56;
    }
    if (Doubles[57] > Split) {
        m |= 1ui64 << 57;
    }
    if (Doubles[58] > Split) {
        m |= 1ui64 << 58;
    }
    if (Doubles[59] > Split) {
        m |= 1ui64 << 59;
    }
    if (Doubles[60] > Split) {
        m |= 1ui64 << 60;
    }
    if (Doubles[61] > Split) {
        m |= 1ui64 << 61;
    }
    if (Doubles[62] > Split) {
        m |= 1ui64 << 62;
    }
    if (Doubles[63] > Split) {
        m |= 1ui64 << 63;
    }
    *Int64 = m;
}
__global__ void copyFloatsToInt32sKernel(float* Floats, uint32_t* Int32s, float Split) {
    copyFloatsToInt32Func(&Floats[blockIdx.x << 5], &Int32s[blockIdx.x], Split);
}
__global__ void copyDoublesToInt32sKernel(double* Floats, uint32_t* Int32s, double Split) {
    copyDoublesToInt32Func(&Floats[blockIdx.x << 5], &Int32s[blockIdx.x], Split);
}
__global__ void initZeroArrayKernel(float* Array) {
    Array[blockIdx.x] = 0.f;
}
__global__ void initZeroArrayKernel(double* Array) {
    Array[blockIdx.x] = 0.;
}
__global__ void initZeroArrayKernel(uint64_t* Array) {
    Array[blockIdx.x] = 0;
}
__global__ void copyFloatsToInt64sKernel(float* Floats, uint64_t* Int64s, float Split) {
    copyFloatsToInt64Func(&Floats[blockIdx.x << 5], &Int64s[blockIdx.x], Split);
}
__global__ void copyDoublesToInt64sKernel(double* Floats, uint64_t* Int64s, double Split) {
    copyDoublesToInt64Func(&Floats[blockIdx.x << 5], &Int64s[blockIdx.x], Split);
}
__global__ void randomizeArrayKernel(uint64_t* Array, uint32_t ProbabilityOf1, uint64_t Seed) {
    Seed = GetSeedOnKernel(Seed);
    DeviceRandom dr(Seed);
    Array[blockIdx.x] ^= Get64Bits(ProbabilityOf1, AnyRNG<uint64_t>(&dr));
}
__global__ void initRandomArrayKernel(uint64_t* Array, uint32_t ProbabilityOf1, uint64_t Seed) {
    DeviceRandom dr(GetSeedOnKernel(Seed));
    Array[blockIdx.x] = Get64Bits(ProbabilityOf1, AnyRNG<uint64_t>(&dr));
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<float> Array, float Scalar, AnyRNG<uint64_t> RNG) {
    Scalar *= 2.0f;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] += Scalar * (dr.GetF() - 0.5f);
    }
#else
    randomizeArrayKernel<<<Array.size, 1>>>(Array.ptr, Scalar, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<double> Array, double Scalar, AnyRNG<uint64_t> RNG) {
    Scalar *= 2.0;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] += Scalar * (dr.GetD() - 0.5);
}
#else
    randomizeArrayKernel<<<Array.size, 1>>>(Array.ptr, Scalar, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<float> Array, float Scalar, float LowerBound, float UpperBound, AnyRNG<uint64_t> RNG) {
    Scalar *= 2.0f;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        float& p(Array[i]);
        float v = p + Scalar * (dr.GetF() - 0.5f);
        if (v < LowerBound) {
            p = LowerBound;
        }
        else if (v > UpperBound) {
            p = UpperBound;
        }
        else {
            p = v;
        }
    }
#else
    randomizeArrayKernel<<<Array.size, 1>>>(Array.ptr, Scalar, LowerBound, UpperBound, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<double> Array, double Scalar, double LowerBound, double UpperBound, AnyRNG<uint64_t> RNG) {
    Scalar *= 2.0;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        double& p(Array[i]);
        double v = p + Scalar * (dr.GetD() - 0.5);
        if (v < LowerBound) {
            p = LowerBound;
        }
        else if (v > UpperBound) {
            p = UpperBound;
        }
        else {
            p = v;
        }
    }
#else
    randomizeArrayKernel<<<Array.size, 1>>>(Array.ptr, Scalar, LowerBound, UpperBound, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<float> Array, AnyRNG<uint64_t> RNG) {
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = dr.GetF();
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<double> Array, AnyRNG<uint64_t> RNG) {
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = dr.GetD();
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<float> Array, float LowerBound, float UpperBound, AnyRNG<uint64_t> RNG) {
    UpperBound -= LowerBound;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = dr.GetF() * UpperBound + LowerBound;
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, LowerBound, UpperBound, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<double> Array, double LowerBound, double UpperBound, AnyRNG<uint64_t> RNG) {
    UpperBound -= LowerBound;
#if __CUDA_ARCH__
    DeviceRandom dr(RNG());
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = dr.GetD() * UpperBound + LowerBound;
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, LowerBound, UpperBound, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<float> Array) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = 0.f;
    }
#else
    initZeroArrayKernel<<<Array.size, 1>>>(Array.ptr);
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<double> Array) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = 0.;
    }
#else
    initZeroArrayKernel<<<Array.size, 1>>>(Array.ptr);
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<uint8_t> Array) {
#if __CUDA_ARCH__
    uint8_t* ptr = Array.ptr;
    size_t l2 = Array.size >> 3;
    for (size_t i = 0; i < l2; ++i) {
        ((uint64_t*)ptr)[i] = 0;
    }
    ptr += Array.size & ~0b111;
    size_t l3 = Array.size & 0b111;
    for (size_t i = 0; i < l3; ++i) {
        ptr[i] = 0;
    }
#else
    initZeroArrayKernel<<<(Array.size >> 3), 1>>>((uint64_t*)Array.ptr);
    uint64_t zero = 0;
    ThrowIfBad(hipMemcpy(Array.ptr + (Array.size & ~0b111), &zero, Array.size & 0b111, hipMemcpyHostToDevice));
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<uint16_t> Array) {
#if __CUDA_ARCH__
    uint16_t* ptr = Array.ptr;
    size_t l2 = Array.size >> 2;
    for (size_t i = 0; i < l2; ++i) {
        ((uint64_t*)ptr)[i] = 0;
    }
    ptr += Array.size & ~0b11;
    size_t l3 = Array.size & 0b11;
    for (size_t i = 0; i < l3; ++i) {
        ptr[i] = 0;
    }
#else
    initZeroArrayKernel<<<(Array.size >> 2), 1>>>((uint64_t*)Array.ptr);
    uint64_t zero = 0;
    ThrowIfBad(hipMemcpy(Array.ptr + (Array.size & ~0b11), &zero, Array.size & 0b11, hipMemcpyHostToDevice));
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<uint32_t> Array) {
#if __CUDA_ARCH__
    size_t l2 = Array.size >> 1;
    for (size_t i = 0; i < l2; ++i) {
        ((uint64_t*)Array.ptr)[i] = 0;
    }
    if (Array.size & 1) {
        Array.ptr[Array.size & ~1] = 0;
    }
#else
    initZeroArrayKernel<<<(Array.size >> 1), 1>>>((uint64_t*)Array.ptr);
    uint64_t zero = 0;
    ThrowIfBad(hipMemcpy(Array.ptr + (Array.size & ~1), &zero, Array.size & 1, hipMemcpyHostToDevice));
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitZeroArray(Span<uint64_t> Array) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = 0;
    }
#else
    initZeroArrayKernel<<<Array.size, 1>>>(Array.ptr);
#endif
}
__global__ void copyFloatsToBoolsKernel(float* Floats, bool* Bools, float Split) {
    Bools[blockIdx.x] = Floats[blockIdx.x] > Split;
}
__host__ void BrendanCUDA::AI::CopyFloatsToBools(float* Floats, bool* Bools, size_t Length, float Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Length; ++i) {
            Bools[i] = Floats[i] > Split;
        }
    }
    else {
        copyFloatsToBoolsKernel<<<Length, 1>>>(Floats, Bools, Split);
    }
}
__global__ void copyDoublesToBoolsKernel(float* Doubles, bool* Bools, float Split) {
    Bools[blockIdx.x] = Doubles[blockIdx.x] > Split;
}
__host__ void BrendanCUDA::AI::CopyDoublesToBools(float* Doubles, bool* Bools, size_t Length, float Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Length; ++i) {
            Bools[i] = Doubles[i] > Split;
        }
    }
    else {
        copyDoublesToBoolsKernel<<<Length, 1>>>(Doubles, Bools, Split);
    }
}
__device__ void BrendanCUDA::AI::CopyFloatsToBools(float* Floats, bool* Bools, size_t Length, float Split) {
    for (size_t i = 0; i < Length; ++i) {
        Bools[i] = Floats[i] > Split;
    }
}
__device__ void BrendanCUDA::AI::CopyDoublesToBools(double* Doubles, bool* Bools, size_t Length, double Split) {
    for (size_t i = 0; i < Length; ++i) {
        Bools[i] = Doubles[i] > Split;
    }
}
__host__ void BrendanCUDA::AI::CopyFloatsToInt32s(float* Floats, uint32_t* Int32s, size_t Int32Length, float Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Int32Length; ++i) {
            copyFloatsToInt32Func(&Floats[i << 5], &Int32s[i], Split);
        }
    }
    else {
        copyFloatsToInt32sKernel<<<Int32Length, 1>>>(Floats, Int32s, Split);
    }
}
__host__ void BrendanCUDA::AI::CopyDoublesToInt32s(double* Doubles, uint32_t* Int32s, size_t Int32Length, double Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Int32Length; ++i) {
            copyDoublesToInt32Func(&Doubles[i << 5], &Int32s[i], Split);
        }
    }
    else {
        copyDoublesToInt32sKernel<<<Int32Length, 1>>>(Doubles, Int32s, Split);
    }
}
__device__ void BrendanCUDA::AI::CopyFloatsToInt32s(float* Floats, uint32_t* Int32s, size_t Int32Length, float Split) {
    for (size_t i = 0; i < Int32Length; ++i) {
        copyFloatsToInt32Func(&Floats[i << 5], &Int32s[i], Split);
    }
}
__device__ void BrendanCUDA::AI::CopyDoublesToInt32s(double* Doubles, uint32_t* Int32s, size_t Int32Length, double Split) {
    for (size_t i = 0; i < Int32Length; ++i) {
        copyDoublesToInt32Func(&Doubles[i << 5], &Int32s[i], Split);
    }
}
__host__ void BrendanCUDA::AI::CopyFloatsToInt64s(float* Floats, uint64_t* Int64s, size_t Int64Length, float Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Int64Length; ++i) {
            copyFloatsToInt64Func(&Floats[i << 5], &Int64s[i], Split);
        }
    }
    else {
        copyFloatsToInt64sKernel<<<Int64Length, 1>>>(Floats, Int64s, Split);
    }
}
__host__ void BrendanCUDA::AI::CopyDoublesToInt64s(double* Doubles, uint64_t* Int64s, size_t Int64Length, double Split, bool MemoryOnHost) {
    if (MemoryOnHost) {
        for (size_t i = 0; i < Int64Length; ++i) {
            copyDoublesToInt64Func(&Doubles[i << 5], &Int64s[i], Split);
        }
    }
    else {
        copyDoublesToInt64sKernel<<<Int64Length, 1>>>(Doubles, Int64s, Split);
    }
}
__device__ void BrendanCUDA::AI::CopyFloatsToInt64s(float* Floats, uint64_t* Int64s, size_t Int64Length, float Split) {
    for (size_t i = 0; i < Int64Length; ++i) {
        copyFloatsToInt64Func(&Floats[i << 5], &Int64s[i], Split);
    }
}
__device__ void BrendanCUDA::AI::CopyDoublesToInt64s(double* Doubles, uint64_t* Int64s, size_t Int64Length, double Split) {
    for (size_t i = 0; i < Int64Length; ++i) {
        copyDoublesToInt64Func(&Doubles[i << 5], &Int64s[i], Split);
    }
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint8_t> Array, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 3;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = RNG();
    }
    if (Array.size & 7) {
        uint64_t n = RNG();
        uint8_t* p = Array.ptr + (l64 << 3);
        switch (Array.size & 7) {
        case 1:
            *p = *(uint8_t*)&n;
            break;
        case 2:
            *(uint16_t*)p = *(uint16_t*)&n;
            break;
        case 3:
            *(uint16_t*)p = *(uint16_t*)&n;
            p[2] = ((uint8_t*)&n)[2];
            break;
        case 4:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 5:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint8_t*)&n)[4];
            break;
        case 6:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            break;
        case 7:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            p[6] = ((uint8_t*)&n)[6];
            break;
        }
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, RNG());
    if (Array.size & 7) {
        uint64_t rv = RNG();
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 3), &rv, (Array.size & 7) * sizeof(uint8_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint16_t> Array, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 2;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = RNG();
    }
    if (Array.size & 3) {
        uint64_t n = RNG();
        uint16_t* p = Array.ptr + (l64 << 2);
        switch (Array.size & 3) {
        case 1:
            *p = *(uint16_t*)&n;
            break;
        case 2:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 3:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[2] = ((uint16_t*)&n)[2];
            break;
        }
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, RNG());
    if (Array.size & 3) {
        uint64_t rv = RNG();
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 2), &rv, (Array.size & 3) * sizeof(uint16_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint32_t> Array, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 1;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = RNG();
    }
    if (Array.size & 1) {
        Array[Array.size - 1] = (uint32_t)RNG();
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, RNG());
    if (Array.size & 1) {
        uint64_t rv = RNG();
        ThrowIfBad(hipMemcpy(Array.ptr + (Array.size - 1), &rv, sizeof(uint32_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint64_t> Array, AnyRNG<uint64_t> RNG) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = RNG();
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<uint64_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] ^= Random::Get64Bits(ProbabilityOf1, RNG);
    }
#else
    randomizeArrayKernel<<<Array.size, 1>>>(Array.ptr, ProbabilityOf1, RNG());
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<uint32_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 1;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 1) {
        Array[Array.size - 1] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
#else
    randomizeArrayKernel<<<l64, 1>>>(a64, ProbabilityOf1, RNG());
    if (Array.size & 1) {
        uint32_t n = (uint32_t)Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (Array.size - 1), &n, sizeof(uint32_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<uint16_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 2;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 3) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        uint16_t* p = Array.ptr + (l64 << 2);
        switch (Array.size & 3) {
        case 1:
            *p = *(uint16_t*)&n;
            break;
        case 2:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 3:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[2] = ((uint16_t*)&n)[2];
            break;
        }
    }
#else
    randomizeArrayKernel<<<l64, 1>>>(a64, ProbabilityOf1, RNG());
    if (Array.size & 3) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 2), &n, (Array.size & 3) * sizeof(uint16_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::RandomizeArray(Span<uint8_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 3;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 7) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        uint8_t* p = Array.ptr + (l64 << 3);
        switch (Array.size & 7) {
        case 1:
            *p = *(uint8_t*)&n;
            break;
        case 2:
            *(uint16_t*)p = *(uint16_t*)&n;
            break;
        case 3:
            *(uint16_t*)p = *(uint16_t*)&n;
            p[2] = ((uint8_t*)&n)[2];
            break;
        case 4:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 5:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint8_t*)&n)[4];
            break;
        case 6:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            break;
        case 7:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            p[6] = ((uint8_t*)&n)[6];
            break;
        }
    }
#else
    randomizeArrayKernel<<<l64, 1>>>(a64, ProbabilityOf1, RNG());
    if (Array.size & 7) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 3), &n, (Array.size & 7) * sizeof(uint8_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint8_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 3;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 7) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        uint8_t* p = Array.ptr + (l64 << 3);
        switch (Array.size & 7) {
        case 1:
            *p = *(uint8_t*)&n;
            break;
        case 2:
            *(uint16_t*)p = *(uint16_t*)&n;
            break;
        case 3:
            *(uint16_t*)p = *(uint16_t*)&n;
            p[2] = ((uint8_t*)&n)[2];
            break;
        case 4:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 5:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint8_t*)&n)[4];
            break;
        case 6:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            break;
        case 7:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[4] = ((uint16_t*)&n)[4];
            p[6] = ((uint8_t*)&n)[6];
            break;
        }
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, RNG());
    if (Array.size & 7) {
        uint64_t rv = Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 3), &rv, (Array.size & 7) * sizeof(uint8_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint16_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 2;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 3) {
        uint64_t n = Random::Get64Bits(ProbabilityOf1, RNG);
        uint16_t* p = Array.ptr + (l64 << 2);
        switch (Array.size & 3) {
        case 1:
            *p = *(uint16_t*)&n;
            break;
        case 2:
            *(uint32_t*)p = *(uint32_t*)&n;
            break;
        case 3:
            *(uint32_t*)p = *(uint32_t*)&n;
            p[2] = ((uint16_t*)&n)[2];
            break;
        }
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, ProbabilityOf1, RNG());
    if (Array.size & 3) {
        uint64_t rv = Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (l64 << 2), &rv, (Array.size & 3) * sizeof(uint16_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint32_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
    size_t l64 = Array.size >> 1;
    uint64_t* a64 = (uint64_t*)Array.ptr;
#if __CUDA_ARCH__
    for (size_t i = 0; i < l64; ++i) {
        a64[i] = Random::Get64Bits(ProbabilityOf1, RNG);
    }
    if (Array.size & 1) {
        Array[Array.size - 1] = (uint32_t)Random::Get64Bits(ProbabilityOf1, RNG);
    }
#else
    initRandomArrayKernel<<<l64, 1>>>(a64, ProbabilityOf1, RNG());
    if (Array.size & 1) {
        uint64_t rv = Random::Get64Bits(ProbabilityOf1, RNG);
        ThrowIfBad(hipMemcpy(Array.ptr + (Array.size - 1), &rv, sizeof(uint32_t), hipMemcpyHostToDevice));
    }
#endif
}
__host__ __device__ void BrendanCUDA::AI::InitRandomArray(Span<uint64_t> Array, uint32_t ProbabilityOf1, AnyRNG<uint64_t> RNG) {
#if __CUDA_ARCH__
    for (size_t i = 0; i < Array.size; ++i) {
        Array[i] = RNG();
    }
#else
    initRandomArrayKernel<<<Array.size, 1>>>(Array.ptr, ProbabilityOf1, RNG());
#endif
}