#include "hip/hip_runtime.h"
#include "brendancuda_nets_net.cuh"
#include <>
#include "brendancuda_cudaerrorhelpers.h"

__host__ __device__ BrendanCUDA::Nets::NetNode::NetNode() {
    data = 0;
    inputs = 0;
    inputCount = 0;
    outputs = 0;
    outputCount = 0;
}

__host__ __device__ void BrendanCUDA::Nets::NetNode::Dispose(dataDestructor_t DataDestructor) {
    if (DataDestructor) {
        DataDestructor(data);
    }
#if __CUDA_ARCH__
    delete[] inputs;
    delete[] outputs;
#else
    hipFree(inputs);
    hipFree(outputs);
#endif
}

BrendanCUDA::Nets::Net::Net() 
    : nodes(*(new thrust::device_vector<NetNode>())) {}

__global__ void disposeKernel(BrendanCUDA::Nets::NetNode* nodes, BrendanCUDA::Nets::dataDestructor_t DataDestructor) {
    nodes[blockIdx.x].Dispose(DataDestructor);
}
void BrendanCUDA::Nets::Net::Dispose(dataDestructor_t DataDestructor) {
    disposeKernel<<<nodes.size(), 1>>>(nodes.data().get(), DataDestructor);
    delete (&nodes);
}

thrust::device_ptr<BrendanCUDA::Nets::NetNode> BrendanCUDA::Nets::Net::Data() {
    return nodes.data();
}

thrust::device_reference<BrendanCUDA::Nets::NetNode> BrendanCUDA::Nets::Net::operator[](size_t i) {
    return nodes[i];
}

__global__ void addConnection_checkForPreexistence(size_t* arr, size_t v, bool* opt) {
    if (arr[blockIdx.x] == v) {
        *opt = true;
    }
}

bool BrendanCUDA::Nets::Net::AddConnection_OnlyInput(size_t InputIndex, size_t OutputIndex, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode in = nodes[InputIndex];

    if (CheckForPreexistence) {
        bool f = false;
        bool* opt;
        ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
        ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
        addConnection_checkForPreexistence<<<in.outputCount, 1>>>(in.outputs, OutputIndex, opt);
        ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
        ThrowIfBad(hipFree(opt));
        if (f) {
            return false;
        }
    }

    bool in_o_e;
    if (CheckForAvailableExcess) {
        size_t s;
        ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(in.outputs)));
        in_o_e = (s >= (in.outputCount + 1) * sizeof(size_t));
    }
    else {
        in_o_e = false;
    }

    if (in_o_e) {
        ThrowIfBad(hipMemcpy(in.outputs + in.outputCount, &OutputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        in.outputCount++;
    }
    else {
        size_t* n;
        size_t noc = in.outputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(size_t) * noc));
        ThrowIfBad(hipMemcpy(n, in.outputs, sizeof(size_t) * in.outputCount, hipMemcpyDeviceToDevice));
        ThrowIfBad(hipMemcpy(n + in.outputCount, &OutputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(in.outputs));
        in.outputs = n;
        in.outputCount = noc;
    }

    nodes[InputIndex] = in;
    return true;
}
bool BrendanCUDA::Nets::Net::AddConnection_OnlyOutput(size_t InputIndex, size_t OutputIndex, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode on = nodes[OutputIndex];

    if (CheckForPreexistence) {
        bool f = false;
        bool* opt;
        ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
        ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
        addConnection_checkForPreexistence<<<on.inputCount, 1>>>(on.inputs, InputIndex, opt);
        ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
        ThrowIfBad(hipFree(opt));
        if (f) {
            return false;
        }
    }

    bool on_i_e;
    if (CheckForAvailableExcess) {
        size_t s;
        ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(on.inputs)));
        on_i_e = (s >= (on.inputCount + 1) * sizeof(size_t));
    }
    else {
        on_i_e = false;
    }

    if (on_i_e) {
        ThrowIfBad(hipMemcpy(on.inputs + on.inputCount, &InputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        on.inputCount++;
    }
    else {
        size_t* n;
        size_t nic = on.inputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(size_t) * nic));
        ThrowIfBad(hipMemcpy(n, on.inputs, sizeof(size_t) * on.inputCount, hipMemcpyDeviceToDevice));
        ThrowIfBad(hipMemcpy(n + on.inputCount, &InputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(on.inputs));
        on.inputs = n;
        on.inputCount = nic;
    }

    nodes[OutputIndex] = on;
    return true;
}
bool BrendanCUDA::Nets::Net::AddConnection(size_t InputIndex, size_t OutputIndex, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode in = nodes[InputIndex];
    NetNode on = nodes[OutputIndex];

    if (CheckForPreexistence) {
        bool f = false;
        bool* opt;
        ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
        ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
        addConnection_checkForPreexistence<<<in.outputCount, 1>>>(in.outputs, OutputIndex, opt);
        ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
        ThrowIfBad(hipFree(opt));
        if (f) {
            return false;
        }
    }

    bool in_o_e;
    bool on_i_e;
    if (CheckForAvailableExcess) {
        size_t s;
        ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(in.outputs)));
        in_o_e = (s >= (in.outputCount + 1) * sizeof(size_t));
        ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(on.inputs)));
        on_i_e = (s >= (on.inputCount + 1) * sizeof(size_t));
    }
    else {
        in_o_e = false;
        on_i_e = false;
    }

    if (in_o_e) {
        ThrowIfBad(hipMemcpy(in.outputs + in.outputCount, &OutputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        in.outputCount++;
    }
    else {
        size_t* n;
        size_t noc = in.outputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(size_t) * noc));
        ThrowIfBad(hipMemcpy(n, in.outputs, sizeof(size_t) * in.outputCount, hipMemcpyDeviceToDevice));
        ThrowIfBad(hipMemcpy(n + in.outputCount, &OutputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(in.outputs));
        in.outputs = n;
        in.outputCount = noc;
    }

    if (on_i_e) {
        ThrowIfBad(hipMemcpy(on.inputs + on.inputCount, &InputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        on.inputCount++;
    }
    else {
        size_t* n;
        size_t nic = on.inputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(size_t) * nic));
        ThrowIfBad(hipMemcpy(n, on.inputs, sizeof(size_t) * on.inputCount, hipMemcpyDeviceToDevice));
        ThrowIfBad(hipMemcpy(n + on.inputCount, &InputIndex, sizeof(size_t), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(on.inputs));
        on.inputs = n;
        on.inputCount = nic;
    }

    nodes[InputIndex] = in;
    nodes[OutputIndex] = on;
    return true;
}

bool BrendanCUDA::Nets::Net::RemoveConnection_OnlyInput(size_t InputIndex, size_t OutputIndex, bool RemoveExcess) {
    NetNode in = nodes[InputIndex];

    size_t* in_o = new size_t[in.outputCount];

    ThrowIfBad(hipMemcpy(in_o, in.outputs, sizeof(size_t) * in.outputCount, hipMemcpyDeviceToHost));

    if (RemoveExcess) {
        for (size_t i = 0; i < in.outputCount; ++i) {
            if (in_o[i] == OutputIndex) {
                in_o[i] = in_o[in.outputCount - 1];
                goto ExitA;
            }
        }
        return false;

    ExitA:
        in.outputCount--;

        size_t* in_o_n;

        ThrowIfBad(hipMalloc(&in_o_n, sizeof(size_t) * in.outputCount));

        ThrowIfBad(hipMemcpy(in_o_n, in_o, sizeof(size_t) * in.outputCount, hipMemcpyHostToDevice));

        delete[] in_o;

        ThrowIfBad(hipFree(in.outputs));

        in.outputs = in_o_n;

        nodes[InputIndex] = in;
    }
    else {
        for (size_t i = 0; i < in.outputCount; ++i) {
            if (in_o[i] == OutputIndex) {
                hipMemcpy(in.outputs + i, in.outputs + (in.outputCount - 1), sizeof(size_t), hipMemcpyDeviceToDevice);
                in_o[i] = in_o[in.outputCount - 1];
                goto ExitB;
            }
        }
        return false;

    ExitB:
        in.outputCount--;
        nodes[InputIndex] = in;
    }
    return true;
}
bool BrendanCUDA::Nets::Net::RemoveConnection_OnlyOutput(size_t InputIndex, size_t OutputIndex, bool RemoveExcess) {
    NetNode on = nodes[OutputIndex];

    size_t* on_i = new size_t[on.inputCount];

    ThrowIfBad(hipMemcpy(on_i, on.inputs, sizeof(size_t) * on.inputCount, hipMemcpyDeviceToHost));

    if (RemoveExcess) {
        for (size_t i = 0; i < on.inputCount; ++i) {
            if (on_i[i] == InputIndex) {
                on_i[i] = on_i[on.inputCount - 1];
                goto ExitA;
            }
        }
        return false;

    ExitA:
        on.inputCount--;

        size_t* on_i_n;

        ThrowIfBad(hipMalloc(&on_i_n, sizeof(size_t) * on.inputCount));

        ThrowIfBad(hipMemcpy(on_i_n, on_i, sizeof(size_t) * on.inputCount, hipMemcpyHostToDevice));

        delete[] on_i;

        ThrowIfBad(hipFree(on.inputs));

        on.inputs = on_i_n;

        nodes[OutputIndex] = on;
    }
    else {
        for (size_t i = 0; i < on.inputCount; ++i) {
            if (on_i[i] == InputIndex) {
                hipMemcpy(on.inputs + i, on.inputs + (on.inputCount - 1), sizeof(size_t), hipMemcpyDeviceToDevice);
                on_i[i] = on_i[on.inputCount - 1];
                goto ExitB;
            }
        }
        return false;

    ExitB:
        on.inputCount--;
        nodes[OutputIndex] = on;
    }
    return true;
}
bool BrendanCUDA::Nets::Net::RemoveConnection(size_t InputIndex, size_t OutputIndex, bool RemoveExcess) {
    NetNode in = nodes[InputIndex];
    NetNode on = nodes[OutputIndex];

    size_t* in_o = new size_t[in.outputCount];
    size_t* on_i = new size_t[on.inputCount];

    ThrowIfBad(hipMemcpy(in_o, in.outputs, sizeof(size_t) * in.outputCount, hipMemcpyDeviceToHost));
    ThrowIfBad(hipMemcpy(on_i, on.inputs, sizeof(size_t) * on.inputCount, hipMemcpyDeviceToHost));

    if (RemoveExcess) {
        for (size_t i = 0; i < in.outputCount; ++i) {
            if (in_o[i] == OutputIndex) {
                in_o[i] = in_o[in.outputCount - 1];
                goto Exit0A;
            }
        }
        return false;

Exit0A:
        for (size_t i = 0; i < on.inputCount; ++i) {
            if (on_i[i] == InputIndex) {
                on_i[i] = on_i[on.inputCount - 1];
                goto Exit1A;
            }
        }
        throw std::exception();

Exit1A:
        in.outputCount--;
        on.inputCount--;

        size_t* in_o_n;
        size_t* on_i_n;

        ThrowIfBad(hipMalloc(&in_o_n, sizeof(size_t) * in.outputCount));
        ThrowIfBad(hipMalloc(&on_i_n, sizeof(size_t) * on.inputCount));

        ThrowIfBad(hipMemcpy(in_o_n, in_o, sizeof(size_t) * in.outputCount, hipMemcpyHostToDevice));
        ThrowIfBad(hipMemcpy(on_i_n, on_i, sizeof(size_t) * on.inputCount, hipMemcpyHostToDevice));

        delete[] in_o;
        delete[] on_i;

        ThrowIfBad(hipFree(in.outputs));
        ThrowIfBad(hipFree(on.inputs));

        in.outputs = in_o_n;
        on.inputs = on_i_n;

        nodes[InputIndex] = in;
        nodes[OutputIndex] = on;
    }
    else {
        for (size_t i = 0; i < in.outputCount; ++i) {
            if (in_o[i] == OutputIndex) {
                hipMemcpy(in.outputs + i, in.outputs + (in.outputCount - 1), sizeof(size_t), hipMemcpyDeviceToDevice);
                in_o[i] = in_o[in.outputCount - 1];
                goto Exit0B;
            }
        }
        return false;

Exit0B:
        for (size_t i = 0; i < on.inputCount; ++i) {
            if (on_i[i] == InputIndex) {
                hipMemcpy(on.inputs + i, on.inputs + (on.inputCount - 1), sizeof(size_t), hipMemcpyDeviceToDevice);
                on_i[i] = on_i[on.inputCount - 1];
                goto Exit1B;
            }
        }
        throw std::exception();

Exit1B:
        in.outputCount--;
        on.inputCount--;
        nodes[InputIndex] = in;
        nodes[OutputIndex] = on;
    }
    return true;
}

void BrendanCUDA::Nets::Net::RemoveAt(size_t Index, dataDestructor_t DataDestructor) {
    NetNode nn = nodes[Index];
    size_t* inputs = new size_t[nn.inputCount];
    size_t* outputs = new size_t[nn.outputCount];
    if (hipMemcpy(inputs, nn.inputs, sizeof(size_t) * nn.inputCount, hipMemcpyDeviceToHost)) {
        throw std::exception();
    }
    if (hipMemcpy(outputs, nn.outputs, sizeof(size_t) * nn.outputCount, hipMemcpyDeviceToHost)) {
        throw std::exception();
    }
    for (size_t i = 0; i < nn.inputCount; ++i) {
        size_t o = inputs[i];
        if (o != Index) {
            RemoveConnection_OnlyInput(o, Index, false);
        }
    }
    for (size_t i = 0; i < nn.outputCount; ++i) {
        size_t o = outputs[i];
        if (o != Index) {
            RemoveConnection_OnlyOutput(Index, o, false);
        }
    }
    delete[] inputs;
    delete[] outputs;
    nn.Dispose(DataDestructor);
    nodes[Index] = nodes[nodes.size() - 1];
    nodes.pop_back();
}