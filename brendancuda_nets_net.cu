#include "hip/hip_runtime.h"
#include "brendancuda_nets_net.cuh"
#include <>
#include "brendancuda_cudaerrorhelpers.h"
#include "brendancuda_crossassignment.h"

__host__ __device__ BrendanCUDA::Nets::NetNode::NetNode() {
    data = 0;
    inputs = 0;
    inputCount = 0;
    outputs = 0;
    outputCount = 0;
}

__host__ __device__ void BrendanCUDA::Nets::NetNode::Dispose(dataDestructor_t DataDestructor) {
    if (DataDestructor) {
        DataDestructor(data);
    }
#if __CUDA_ARCH__
    delete[] inputs;
    delete[] outputs;
#else
    ThrowIfBad(hipFree(inputs));
    ThrowIfBad(hipFree(outputs));
#endif
}

BrendanCUDA::Nets::Net::Net() 
    : nodes(*(new thrust::device_vector<NetNode>())) {}

__global__ void disposeKernel(BrendanCUDA::Nets::NetNode* nodes, BrendanCUDA::Nets::dataDestructor_t DataDestructor) {
    nodes[blockIdx.x].Dispose(DataDestructor);
}
void BrendanCUDA::Nets::Net::Dispose(dataDestructor_t DataDestructor) {
    disposeKernel<<<nodes.size(), 1>>>(nodes.data().get(), DataDestructor);
    delete (&nodes);
}

thrust::device_vector<BrendanCUDA::Nets::NetNode>& BrendanCUDA::Nets::Net::DataVec() const {
    return nodes;
}
thrust::device_ptr<BrendanCUDA::Nets::NetNode> BrendanCUDA::Nets::Net::DataPtr() const {
    return nodes.data();
}

thrust::device_reference<BrendanCUDA::Nets::NetNode> BrendanCUDA::Nets::Net::operator[](size_t i) const {
    return nodes[i];
}

__global__ void net_addConnection_checkForPreexistence(BrendanCUDA::Nets::NetNode** arr, BrendanCUDA::Nets::NetNode* v, bool* opt) {
    if (arr[blockIdx.x] == v) {
        *opt = true;
    }
}

bool BrendanCUDA::Nets::Net::AddConnection_OnlyInput(NetNode* InputNode, NetNode* OutputNode, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode in = GetVR(InputNode);

    if (in.outputs) {
        if (CheckForPreexistence) {
            bool f = false;
            bool* opt;
            ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
            ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
            net_addConnection_checkForPreexistence<<<in.outputCount, 1>>>(in.outputs, OutputNode, opt);
            ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
            ThrowIfBad(hipFree(opt));
            if (f) {
                return false;
            }
        }

        bool in_o_e;
        if (CheckForAvailableExcess) {
            size_t s;
            ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(in.outputs)));
            in_o_e = (s >= (in.outputCount + 1) * sizeof(NetNode*));
        }
        else {
            in_o_e = false;
        }

        if (in_o_e) {
            ThrowIfBad(hipMemcpy(in.outputs + in.outputCount, &OutputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
            in.outputCount++;
        }
        else {
            NetNode** n;
            size_t noc = in.outputCount + 1;
            ThrowIfBad(hipMalloc(&n, sizeof(NetNode*) * noc));
            ThrowIfBad(hipMemcpy(n, in.outputs, sizeof(NetNode*) * in.outputCount, hipMemcpyDeviceToDevice));
            ThrowIfBad(hipMemcpy(n + in.outputCount, &OutputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
            ThrowIfBad(hipFree(in.outputs));
            in.outputs = n;
            in.outputCount = noc;
        }
    }
    else {
        ThrowIfBad(hipMalloc(&in.outputs, sizeof(NetNode*)));
        SetVR(in.outputs, OutputNode);
        in.outputCount = 1;
    }

    SetVR(InputNode, in);
    return true;
}
bool BrendanCUDA::Nets::Net::AddConnection_OnlyOutput(NetNode* InputNode, NetNode* OutputNode, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode on = GetVR(InputNode);

    if (on.inputs) {
        if (CheckForPreexistence) {
            bool f = false;
            bool* opt;
            ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
            ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
            net_addConnection_checkForPreexistence<<<on.inputCount, 1>>>(on.inputs, InputNode, opt);
            ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
            ThrowIfBad(hipFree(opt));
            if (f) {
                return false;
            }
        }

        bool on_i_e;
        if (CheckForAvailableExcess) {
            size_t s;
            ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(on.inputs)));
            on_i_e = (s >= (on.inputCount + 1) * sizeof(NetNode*));
        }
        else {
            on_i_e = false;
        }

        if (on_i_e) {
            ThrowIfBad(hipMemcpy(on.inputs + on.inputCount, &InputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
            on.inputCount++;
        }
        else {
            NetNode** n;
            size_t nic = on.inputCount + 1;
            ThrowIfBad(hipMalloc(&n, sizeof(NetNode*) * nic));
            ThrowIfBad(hipMemcpy(n, on.inputs, sizeof(NetNode*) * on.inputCount, hipMemcpyDeviceToDevice));
            ThrowIfBad(hipMemcpy(n + on.inputCount, &InputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
            ThrowIfBad(hipFree(on.inputs));
            on.inputs = n;
            on.inputCount = nic;
        }
    }
    else {
        ThrowIfBad(hipMalloc(&on.inputs, sizeof(NetNode*)));
        SetVR(on.inputs, InputNode);
        on.inputCount = 1;
    }

    SetVR(OutputNode, on);
    return true;
}
bool BrendanCUDA::Nets::Net::AddConnection(NetNode* InputNode, NetNode* OutputNode, bool CheckForPreexistence, bool CheckForAvailableExcess) {
    NetNode in = GetVR(InputNode);
    NetNode on = GetVR(OutputNode);

    if (CheckForPreexistence && in.outputs) {
        bool f = false;
        bool* opt;
        ThrowIfBad(hipMalloc(&opt, sizeof(bool)));
        ThrowIfBad(hipMemcpy(opt, &f, sizeof(bool), hipMemcpyHostToDevice));
        net_addConnection_checkForPreexistence<<<in.outputCount, 1>>>(in.outputs, OutputNode, opt);
        ThrowIfBad(hipMemcpy(&f, opt, sizeof(bool), hipMemcpyDeviceToHost));
        ThrowIfBad(hipFree(opt));
        if (f) {
            return false;
        }
    }

    bool in_o_e;
    bool on_i_e;
    if (CheckForAvailableExcess) {
        size_t s;
        if (in.outputs) {
            ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(in.outputs)));
            in_o_e = (s >= (in.outputCount + 1) * sizeof(NetNode*));
        }
        else {
            in_o_e = false;
        }
        if (on.inputs) {
            ThrowIfBad(hipGetSymbolSize(&s, HIP_SYMBOL(on.inputs)));
            on_i_e = (s >= (on.inputCount + 1) * sizeof(NetNode*));
        }
        else {
            on_i_e = false;
        }
    }
    else {
        in_o_e = false;
        on_i_e = false;
    }

    if (in.outputs) {
        ThrowIfBad(hipMalloc(&in.outputs, sizeof(NetNode*)));
        SetVR(in.outputs, OutputNode);
        in.outputCount = 1;
    }
    else if (in_o_e) {
        ThrowIfBad(hipMemcpy(in.outputs + in.outputCount, &OutputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
        in.outputCount++;
    }
    else {
        NetNode** n;
        size_t noc = in.outputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(NetNode*) * noc));
        if (in.outputs) {
            ThrowIfBad(hipMemcpy(n, in.outputs, sizeof(NetNode*) * in.outputCount, hipMemcpyDeviceToDevice));
        }
        ThrowIfBad(hipMemcpy(n + in.outputCount, &OutputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(in.outputs));
        in.outputs = n;
        in.outputCount = noc;
    }

    if (on.inputs) {
        ThrowIfBad(hipMalloc(&on.inputs, sizeof(NetNode*)));
        SetVR(on.inputs, InputNode);
        on.inputCount = 1;
    }
    else if (on_i_e) {
        ThrowIfBad(hipMemcpy(on.inputs + on.inputCount, &InputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
        on.inputCount++;
    }
    else {
        NetNode** n;
        size_t nic = on.inputCount + 1;
        ThrowIfBad(hipMalloc(&n, sizeof(NetNode*) * nic));
        if (on.inputs) {
            ThrowIfBad(hipMemcpy(n, on.inputs, sizeof(NetNode*) * on.inputCount, hipMemcpyDeviceToDevice));
        }
        ThrowIfBad(hipMemcpy(n + on.inputCount, &InputNode, sizeof(NetNode*), hipMemcpyHostToDevice));
        ThrowIfBad(hipFree(on.inputs));
        on.inputs = n;
        on.inputCount = nic;
    }

    SetVR(InputNode, in);
    SetVR(OutputNode, on);
    return true;
}

bool BrendanCUDA::Nets::Net::RemoveConnection_OnlyInput(NetNode* InputNode, NetNode* OutputNode, bool RemoveExcess) {
    NetNode in = GetVR(InputNode);

    if (in.outputs) {
        NetNode** in_o = new NetNode * [in.outputCount];

        ThrowIfBad(hipMemcpy(in_o, in.outputs, sizeof(NetNode*) * in.outputCount, hipMemcpyDeviceToHost));

        if (RemoveExcess) {
            for (size_t i = 0; i < in.outputCount; ++i) {
                if (in_o[i] == OutputNode) {
                    in_o[i] = in_o[in.outputCount - 1];
                    goto ExitA;
                }
            }
            return false;

        ExitA:
            in.outputCount--;

            NetNode** in_o_n;

            ThrowIfBad(hipMalloc(&in_o_n, sizeof(NetNode*) * in.outputCount));

            ThrowIfBad(hipMemcpy(in_o_n, in_o, sizeof(NetNode*) * in.outputCount, hipMemcpyHostToDevice));

            delete[] in_o;

            ThrowIfBad(hipFree(in.outputs));

            in.outputs = in_o_n;

            SetVR(InputNode, in);
        }
        else {
            for (size_t i = 0; i < in.outputCount; ++i) {
                if (in_o[i] == OutputNode) {
                    ThrowIfBad(hipMemcpy(in.outputs + i, in.outputs + (in.outputCount - 1), sizeof(NetNode*), hipMemcpyDeviceToDevice));
                    in_o[i] = in_o[in.outputCount - 1];
                    goto ExitB;
                }
            }
            return false;

        ExitB:
            in.outputCount--;
            SetVR(InputNode, in);
        }
        return true;
    }
    else {
        return false;
    }
}
bool BrendanCUDA::Nets::Net::RemoveConnection_OnlyOutput(NetNode* InputNode, NetNode* OutputNode, bool RemoveExcess) {
    NetNode on = GetVR(OutputNode);

    if (on.inputs) {
        NetNode** on_i = new NetNode * [on.inputCount];

        ThrowIfBad(hipMemcpy(on_i, on.inputs, sizeof(NetNode*) * on.inputCount, hipMemcpyDeviceToHost));

        if (RemoveExcess) {
            for (size_t i = 0; i < on.inputCount; ++i) {
                if (on_i[i] == InputNode) {
                    on_i[i] = on_i[on.inputCount - 1];
                    goto ExitA;
                }
            }
            return false;

        ExitA:
            on.inputCount--;

            NetNode** on_i_n;

            ThrowIfBad(hipMalloc(&on_i_n, sizeof(NetNode*) * on.inputCount));

            ThrowIfBad(hipMemcpy(on_i_n, on_i, sizeof(NetNode*) * on.inputCount, hipMemcpyHostToDevice));

            delete[] on_i;

            ThrowIfBad(hipFree(on.inputs));

            on.inputs = on_i_n;

            SetVR(OutputNode, on);
        }
        else {
            for (size_t i = 0; i < on.inputCount; ++i) {
                if (on_i[i] == InputNode) {
                    ThrowIfBad(hipMemcpy(on.inputs + i, on.inputs + (on.inputCount - 1), sizeof(NetNode*), hipMemcpyDeviceToDevice));
                    on_i[i] = on_i[on.inputCount - 1];
                    goto ExitB;
                }
            }
            return false;

        ExitB:
            on.inputCount--;
            SetVR(OutputNode, on);
        }
        return true;
    }
    else {
        return false;
    }
}
bool BrendanCUDA::Nets::Net::RemoveConnection(NetNode* InputNode, NetNode* OutputNode, bool RemoveExcess) {
    NetNode in = GetVR(InputNode);
    NetNode on = GetVR(OutputNode);

    if (in.outputs || on.inputs) {
        NetNode** in_o = new NetNode * [in.outputCount];
        NetNode** on_i = new NetNode * [on.inputCount];

        ThrowIfBad(hipMemcpy(in_o, in.outputs, sizeof(NetNode*) * in.outputCount, hipMemcpyDeviceToHost));
        ThrowIfBad(hipMemcpy(on_i, on.inputs, sizeof(NetNode*) * on.inputCount, hipMemcpyDeviceToHost));

        if (RemoveExcess) {
            for (size_t i = 0; i < in.outputCount; ++i) {
                if (in_o[i] == OutputNode) {
                    in_o[i] = in_o[in.outputCount - 1];
                    goto Exit0A;
                }
            }
            return false;

        Exit0A:
            for (size_t i = 0; i < on.inputCount; ++i) {
                if (on_i[i] == InputNode) {
                    on_i[i] = on_i[on.inputCount - 1];
                    goto Exit1A;
                }
            }
            throw std::exception();

        Exit1A:
            in.outputCount--;
            on.inputCount--;

            NetNode** in_o_n;
            NetNode** on_i_n;

            ThrowIfBad(hipMalloc(&in_o_n, sizeof(NetNode*) * in.outputCount));
            ThrowIfBad(hipMalloc(&on_i_n, sizeof(NetNode*) * on.inputCount));

            ThrowIfBad(hipMemcpy(in_o_n, in_o, sizeof(NetNode*) * in.outputCount, hipMemcpyHostToDevice));
            ThrowIfBad(hipMemcpy(on_i_n, on_i, sizeof(NetNode*) * on.inputCount, hipMemcpyHostToDevice));

            delete[] in_o;
            delete[] on_i;

            ThrowIfBad(hipFree(in.outputs));
            ThrowIfBad(hipFree(on.inputs));

            in.outputs = in_o_n;
            on.inputs = on_i_n;

            SetVR(InputNode, in);
            SetVR(OutputNode, on);
        }
        else {
            for (size_t i = 0; i < in.outputCount; ++i) {
                if (in_o[i] == OutputNode) {
                    ThrowIfBad(hipMemcpy(in.outputs + i, in.outputs + (in.outputCount - 1), sizeof(NetNode*), hipMemcpyDeviceToDevice));
                    in_o[i] = in_o[in.outputCount - 1];
                    goto Exit0B;
                }
            }
            return false;

        Exit0B:
            for (size_t i = 0; i < on.inputCount; ++i) {
                if (on_i[i] == InputNode) {
                    ThrowIfBad(hipMemcpy(on.inputs + i, on.inputs + (on.inputCount - 1), sizeof(NetNode*), hipMemcpyDeviceToDevice));
                    on_i[i] = on_i[on.inputCount - 1];
                    goto Exit1B;
                }
            }
            throw std::exception();

        Exit1B:
            in.outputCount--;
            on.inputCount--;
            SetVR(InputNode, in);
            SetVR(OutputNode, on);
        }
        return true;
    }
    else {
        return false;
    }
}

void BrendanCUDA::Nets::Net::RemoveAllConnections(NetNode* Node) {
    NetNode nn = GetVR(Node);
    NetNode** inputs = new NetNode*[nn.inputCount];
    NetNode** outputs = new NetNode*[nn.outputCount];
    ThrowIfBad(hipMemcpy(inputs, nn.inputs, sizeof(NetNode*) * nn.inputCount, hipMemcpyDeviceToHost));
    ThrowIfBad(hipMemcpy(outputs, nn.outputs, sizeof(NetNode*) * nn.outputCount, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < nn.inputCount; ++i) {
        NetNode* o = inputs[i];
        if (o != Node) {
            RemoveConnection_OnlyInput(o, Node, false);
        }
    }
    for (size_t i = 0; i < nn.outputCount; ++i) {
        NetNode* o = outputs[i];
        if (o != Node) {
            RemoveConnection_OnlyOutput(Node, o, false);
        }
    }
    delete[] inputs;
    delete[] outputs;
    ThrowIfBad(hipFree(nn.inputs));
    ThrowIfBad(hipFree(nn.outputs));
    nn.inputCount = 0;
    nn.inputs = 0;
    nn.outputCount = 0;
    nn.outputs = 0;
}